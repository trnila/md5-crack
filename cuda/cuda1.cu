#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include "../crack.h"

#define LETTERS const char letters[] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";
//#define LETTERS const char letters[] = "0123456789";
#define MAX_WORDSIZE 10

#define CHECK(fn) fn; \
	if ( ( cerr = hipGetLastError() ) != hipSuccess ) { \
		printf( "CUDA Error [%d] - '%s'" #fn "\n", __LINE__, hipGetErrorString( cerr ) ); \
		exit(1); \
	}
__device__ void md5(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest);

__device__ bool hash_equals(void* pa, void* pb, int len) {
	int *a = (int*) pa;
	int *b = (int*) pb;
	while(len--) {
		if(*a != *b) {
			return false;
		}
		a++;
		b++;
	}

	return true;
}

__global__ void thread_hierarchy(int len, uint8_t *search, char *table) {
	LETTERS

	int a = blockDim.x * blockIdx.x + threadIdx.x;
	int b = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.z * blockIdx.z + threadIdx.z;

	if(a >= sizeof(letters) || b >= sizeof(letters) || c >= sizeof(letters)) {
		return;
	}

	uint8_t start[MAX_WORDSIZE];
	for(int i = 0; i < MAX_WORDSIZE; i++) {
		start[i] = letters[0];
	}
	start[0] = letters[a];
	start[1] = letters[b];
	start[2] = letters[c];
	start[len] = 0;

	uint8_t hash[16];
	for(;;) {	
		md5(start, len, hash);
		if(hash_equals(hash, search, sizeof(hash))) {
			printf("Hash found %s\n", start);
			__threadfence();
			asm("trap;");
			return;
		}

		int i = len - 1;
		for(;;) {
			// increment last char
			start[i] = table[start[i]];

			// character not overflowed to start, break and test new combination
			if(start[i] != letters[0]) {
				break;
			}

			i--;

			if(i < 3) {
				return;
			}
		}
	}


}

void cuda_crack(int wordLength, uint8_t *hash) {
	LETTERS

	hipError_t cerr;

	//CHECK(hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*1024*80));

	uint8_t *gpu;
	CHECK(hipMalloc(&gpu, 16));
	CHECK(hipMemcpy(gpu, hash, 16, hipMemcpyHostToDevice));

	char table[255];
	char* gpuTable;
	initTable(table);
	CHECK(hipMalloc(&gpuTable, sizeof(table)));
	CHECK(hipMemcpy(gpuTable, table, sizeof(table), hipMemcpyHostToDevice));

	int len = sizeof(letters);
	int threadsSize = 10;
	int blockSize = (len + threadsSize )/ threadsSize;
	thread_hierarchy<<< dim3(blockSize, blockSize, blockSize), dim3(threadsSize, threadsSize, threadsSize)>>>(wordLength, gpu, gpuTable);
	CHECK(1);

	CHECK(hipDeviceSynchronize());
}
