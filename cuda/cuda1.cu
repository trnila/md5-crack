#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include "../crack.h"

#define LETTERS const char letters[] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";
//#define LETTERS const char letters[] = "0123456789";
#define MAX_WORDSIZE 10

#define CHECK(fn) fn; \
	if ( ( cerr = hipGetLastError() ) != hipSuccess ) { \
		printf( "CUDA Error [%d] - '%s'" #fn "\n", __LINE__, hipGetErrorString( cerr ) ); \
		exit(1); \
	}
__device__ void md5_vfy(unsigned char* data, uint length, uint *a1, uint *b1, uint *c1, uint *d1);
void md5_to_ints(unsigned char* md5, uint *r0, uint *r1, uint *r2, uint *r3);
void initTable(char* table) {
    for(int i = 0; i < 254; i++) {
	    table[i] = i + 1;
    }
    table['9'] = 'A';
    table['Z'] = 'a';
    table['z'] = '0';
}

__global__ void thread_hierarchy(int len, char *table, uint32_t ha, uint32_t hb, uint32_t hc, uint32_t hd) {
	LETTERS
	uint32_t search[4];
	search[0] = ha;
	search[1] = hb;
	search[2] = hc;
	search[3] = hd;

	int a = blockDim.x * blockIdx.x + threadIdx.x;
	int b = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.z * blockIdx.z + threadIdx.z;

	if(a >= sizeof(letters) || b >= sizeof(letters) || c >= sizeof(letters)) {
		return;
	}

	uint8_t start[MAX_WORDSIZE];
	for(int i = 0; i < MAX_WORDSIZE; i++) {
		start[i] = letters[0];
	}
	start[0] = letters[a];
	start[1] = letters[b];
	start[2] = letters[c];
	start[len] = 0;

	uint32_t computed[4];
	for(;;) {	
		md5_vfy(start, len, &computed[0], &computed[1], &computed[2], &computed[3]);
		if(computed[0] == search[0] && computed[1] == search[1] && computed[2] == search[2] && computed[3] == search[3]) {
			printf("Hash found %s\n", start);
			__threadfence();
			asm("trap;");
			return;
		}

		int i = len - 1;
		for(;;) {
			// increment last char
			start[i] = table[start[i]];

			// character not overflowed to start, break and test new combination
			if(start[i] != letters[0]) {
				break;
			}

			i--;

			if(i < 3) {
				return;
			}
		}
	}
}

void cuda_crack(int wordLength, uint8_t *hash) {
	LETTERS

	hipError_t cerr;

	//CHECK(hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*1024*80));

	uint32_t search[4];
	md5_to_ints(hash, &search[0], &search[1], &search[2], &search[3]);

	char table[255];
	char* gpuTable;
	initTable(table);
	CHECK(hipMalloc(&gpuTable, sizeof(table)));
	CHECK(hipMemcpy(gpuTable, table, sizeof(table), hipMemcpyHostToDevice));

	int len = sizeof(letters);
	int threadsSize = 5;
	int blockSize = (len + threadsSize ) / threadsSize;
	printf("blocks(%d, %d, %d) threadsInBlock(%d, %d, %d)\n", blockSize, blockSize, blockSize, threadsSize, threadsSize, threadsSize);
	thread_hierarchy<<< dim3(blockSize, blockSize, blockSize), dim3(threadsSize, threadsSize, threadsSize)>>>(wordLength, gpuTable, search[0], search[1], search[2], search[3]);
	CHECK(1);

	CHECK(hipDeviceSynchronize());
}
