#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include "../crack.h"

#define MAX_WORDSIZE 10
#define TOTAL_LETTERS 62

#define CHECK(fn) fn; \
	if ( ( cerr = hipGetLastError() ) != hipSuccess ) { \
		printf( "CUDA Error [%d] - '%s'" #fn "\n", __LINE__, hipGetErrorString( cerr ) ); \
		exit(1); \
	}
__device__ void md5_vfy(unsigned char* data, uint length, uint *a1, uint *b1, uint *c1, uint *d1);
void md5_to_ints(unsigned char* md5, uint *r0, uint *r1, uint *r2, uint *r3);


__device__ char get_letter(int pos) {
	if(pos < 10) {
		return '0' + pos;
	}

	if(pos < 36) {
		return 'a' + pos - 10;
	}	

	return 'A' + pos - 36;
}

__global__ void thread_hierarchy(int len, uint32_t ha, uint32_t hb, uint32_t hc, uint32_t hd) {
	uint32_t search[4];
	search[0] = ha;
	search[1] = hb;
	search[2] = hc;
	search[3] = hd;

	int a = blockDim.x * blockIdx.x + threadIdx.x;
	int b = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.z * blockIdx.z + threadIdx.z;

	if(a >= TOTAL_LETTERS || b >= TOTAL_LETTERS || c >= TOTAL_LETTERS) {
		return;
	}

	uint8_t start[MAX_WORDSIZE];
	for(int i = 0; i < MAX_WORDSIZE; i++) {
		start[i] = '0';
	}
	start[0] = get_letter(a);
	start[1] = get_letter(b);
	start[2] = get_letter(c);
	start[len] = 0;

	uint32_t computed[4];
	int checked = 0;
	for(;;) {	
		checked++;
		md5_vfy(start, len, &computed[0], &computed[1], &computed[2], &computed[3]);
		if(computed[0] == search[0] && computed[1] == search[1] && computed[2] == search[2] && computed[3] == search[3]) {
			printf("Hash found %s\n", start);
			__threadfence();
			asm("trap;");
			return;
		}

		int i = len - 1;
		for(;;) {
			// increment last char
			start[i]++;
			if(i < 3) {
//				printf("%d\n", checked);
				return;
			}
			if(start[i] == ':') {
				start[i] = 'A';
			} else if(start[i] == '[') {
				start[i] = 'a';
			} else if(start[i] == '{') {
				start[i] = '0';
				i--;

				continue;
			}
			break;
		}
	}
}

void cuda_crack(int wordLength, uint8_t *hash) {
	hipError_t cerr;

	CHECK(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
	CHECK(hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*1024*80));

	uint32_t search[4];
	md5_to_ints(hash, &search[0], &search[1], &search[2], &search[3]);

	int len = 62; // total number of characters
	int threadsSize = 5;
	int blockSize = (len + threadsSize ) / threadsSize;
	printf("blocks(%d, %d, %d) threadsInBlock(%d, %d, %d)\n", blockSize, blockSize, blockSize, threadsSize, threadsSize, threadsSize);
	thread_hierarchy<<< dim3(blockSize, blockSize, blockSize), dim3(threadsSize, threadsSize, threadsSize)>>>(wordLength, search[0], search[1], search[2], search[3]);
	CHECK(1);

	CHECK(hipDeviceSynchronize());
}
